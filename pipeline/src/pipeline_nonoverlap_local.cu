#include "hip/hip_runtime.h"
#include "gemmFactory.cuh"
#include "pipeline.h"
#include "spdlog/spdlog.h"

NonOverlapLocalPipeline::NonOverlapLocalPipeline(vortexInitData* input_data,
									   int nrank,
									   int nranks,
									   int vnranks)
	: PipelineBase(input_data, nrank, nranks, vnranks) {
	// sampled tokens 
	hipHostMalloc(&outputTokens, 4096*sizeof(int));
	init();
}

void NonOverlapLocalPipeline::init() {
	stream_all = hipStreamPerThread;
	spdlog::info("Init pipeline (non-overlap local)");
#ifdef ENABLE_NETWORK
	spdlog::info("Init net");
	NetOpPrepare();
#endif
	// CUDA_CHECK(hipMalloc(&weight_buffer, ((size_t)4)*1024*1024*1024));
}

vortexOutputData NonOverlapLocalPipeline::run() {
	constexpr bool enableGraph = false;
	spdlog::info("Start run");
	setWeight(0);

	if(!enableGraph)
		CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_START], stream_all));
	if(enableGraph) hipStreamBeginCapture(stream_all, hipStreamCaptureModeGlobal);

	// TODO: setup phase
	// layerNormPipeStart(stream_gemm);
	// KQV_START->run(stream_gemm);
	// GEMV_START(stream_gemm);
	genEmbedding.run().log(private_logger);

	for (int iter = 0; iter < ModelConfig.run_layer; ++iter) {
		setWeight(iter);
		layerNormAttention.run().log(private_logger);
		KQV->run().log(private_logger);
		roPEAppend.run().log(private_logger);
		GEMV.run().log(private_logger);
		if (update_data.prefillNum > 0)
			prefill.run().log(private_logger);

		O->run().log(private_logger);
		layerNormFFN.run().log(private_logger);
		// UG->run().log(private_logger);
		// activation.run().log(private_logger);
		dual.run().log(private_logger);
		D->run().log(private_logger);
	}
	keepToken.run().log(private_logger);
	layerNormModel.run().log(private_logger);
	LOGITS->run().log(private_logger);
	maxSampler.run().log(private_logger);
	// hipMemcpyAsync(
	// 	outputTokens, input_data->tmp_buffer, 2048 * sizeof(int), hipMemcpyDeviceToHost, stream_all);
	// // End capture
	hipGraph_t graph;
	if (enableGraph) {
		hipStreamEndCapture(stream_all, &graph);
		if (graph == NULL) {
			spdlog::error("Failed to create graph");
			exit(1);
		}
		spdlog::info("Graph created");
		hipGraphExec_t instance;
		hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
		spdlog::info("Graph instantiated");
		CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_START], stream_all));
		for(int i = 0; i < 10; i ++)
			hipGraphLaunch(instance, stream_all);
		spdlog::info("Graph launched");
	}

	// Record an event when the GEMMs are complete
	CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_END], stream_all));

	// Wait for work on the device to complete.
	CUDA_CHECK(hipEventSynchronize(events[EventManager::GEMM_TIMING_END]));

	// Measure elapsed runtime
	float runtime_ms = 0;
	CUDA_CHECK(hipEventElapsedTime(&runtime_ms,
									events[EventManager::GEMM_TIMING_START],
									events[EventManager::GEMM_TIMING_END]));
	// Compute average runtime and GFLOPs.
	runtime_ms = double(runtime_ms);
	spdlog::info("Total running cost (ms) of one microbatch is {}", runtime_ms);

	// Copy output data back to host
	spdlog::info("sampled_token_array {}, maxSampler.d_argMax.ptr {}, sampled_tokens {}", (size_t) output_data.sampled_token_array, (size_t)maxSampler.d_argMax.ptr, output_data.sampled_tokens);
	CUDA_CHECK(hipMemcpy(output_data.sampled_token_array, maxSampler.d_argMax.ptr, output_data.sampled_tokens* sizeof(int), hipMemcpyDeviceToHost));
	
	return output_data;

	// vortexOutputData d;
	// d.global_batch_size = 0;
	// d.sampled_tokens = 0;
	
	// return d;
}

void NonOverlapLocalPipeline::update(vortexUpdateData* update_data_) {
	this->update_data = *update_data_;
	genEmbedding.setInput(pllmTensor<int>(update_data.input_tokens, config_data.global_batch_size, 1, PllmLayout::ROW_MAJOR));
	int req_num = update_data.decodePrefillBorder + update_data.prefillNum;
	maxSampler.set_batch_size(req_num/nranks);
	keepToken.update(req_num, update_data.input_indptr);
	LOGITS->set_shape((req_num/nranks+127)/128*128, ModelConfig.vocab_size, ModelConfig.model_hidden_dim);
	LOGITS->init();
	output_data.sampled_tokens = req_num;
	
	GEMVOpUpdate();
	// log first batch size
	spdlog::info(
		"prefill: {}, decode: {}", update_data.prefillNum, update_data.decodePrefillBorder);
	spdlog::info("Batch size: {}, {}, {}, {}",
				 update_data.gemv_batch_size[0],
				 update_data.gemv_batch_size[1],
				 update_data.gemv_batch_size[2],
				 update_data.gemv_batch_size[3]);
}

void NonOverlapLocalPipeline::config(vortexConfigData* config_data) {
	spdlog::info("Config non-overlap local pipeline");
	this->config_data = * config_data;
    for (int i = 0; i < gemmNum; ++i) {
        gemms[i] = generateGEMM(this->config_data.gemm_op_tag[i]);
		spdlog::info("GEMM {} created, tag: {}", i, this->config_data.gemm_op_tag[i]);
    }
    int globalbatch = config_data->global_batch_size;
    O ->set_shape(globalbatch, ModelConfig.model_hidden_dim_pergpu, ModelConfig.model_hidden_dim);
    
	D ->set_shape(globalbatch, ModelConfig.model_hidden_dim, ModelConfig.model_ff_dim_gpu);
	KQV->set_shape(globalbatch,
				   (ModelConfig.model_kv_heads_gpu + ModelConfig.model_kv_heads_gpu + ModelConfig.model_qo_heads_gpu) * ModelConfig.model_head_dim,
				   ModelConfig.model_hidden_dim);
	LOGITS->set_shape(globalbatch, ModelConfig.vocab_size, ModelConfig.model_hidden_dim);
	dual.set_shape(globalbatch, ModelConfig.model_ff_dim_gpu, ModelConfig.model_hidden_dim);

	spdlog::info("Init schedule");
	ScheduleInit();
	spdlog::info("Init gemm");
	GEMMOpInit();
	spdlog::info("Init gemv");
	GEMVOpInit();
	spdlog::info("Init other");
	OtherOpInit();
	setName();
	// init the output 
	output_data = vortexOutputData();
	output_data.sampled_token_array = new int[config_data->global_batch_size];
	output_data.global_batch_size = config_data->global_batch_size;
}
void NonOverlapLocalPipeline::setName(){
		SET_NAME_PTR(KQV);
		SET_NAME_PTR(LOGITS);
		SET_NAME_PTR(O);
		SET_NAME_REF(dual);
		SET_NAME_PTR(D);

		SET_NAME_REF(genEmbedding);
		SET_NAME_REF(layerNormAttention);
		SET_NAME_REF(layerNormFFN);
		SET_NAME_REF(layerNormModel);
		SET_NAME_REF(GEMV);
		SET_NAME_REF(prefill);
		SET_NAME_REF(roPEAppend);
		SET_NAME_REF(maxSampler);
		SET_NAME_REF(keepToken);
}
void NonOverlapLocalPipeline::setWeight(int layer) {
    bool success = true;
	success &= O->set_weight(input_data->weight.layer_weight[layer].W_O1);

	success &= D->set_weight(input_data->weight.layer_weight[layer].W_D);

	success &= KQV->set_weight(input_data->weight.layer_weight[layer].W_KQV);

	success &= layerNormAttention.setWeight(input_data->weight.layer_weight[layer].W_LN_Attention);
	
	success &= layerNormFFN.setWeight(input_data->weight.layer_weight[layer].W_LN_FFN);
	
	success &= dual.set_weight(input_data->weight.layer_weight[layer].W_G, input_data->weight.layer_weight[layer].W_U);

	if (!success) {
		spdlog::error("Failed to set weight for layer {}", layer);
	}

	GEMV.setKVData(input_data->kv_data[layer]);
	prefill.setKVData(input_data->kv_data[layer]);
	roPEAppend.setKVData(input_data->kv_data[layer]);




	// AllocationManager<cutlass::half_t> weightM((cutlass::half_t*)weight_buffer, ((size_t)4)*1024*1024*1024);

	// const auto& W_embedding = weightM.allocSpan((size_t)ModelConfig.vocab_size * ModelConfig.model_hidden_dim);
	// spdlog::info("embedding location {}", (size_t)W_embedding.data());
	// auto vW_embedding = vortexWeight{(half*)W_embedding.data(), ModelConfig.vocab_size, ModelConfig.model_hidden_dim};
	// genEmbedding.setWeight(vW_embedding);


	// const auto& W_KQV = weightM.allocSpan(KQV->kn());
	// KQV->set_weight(W_KQV.data());
	// const auto& W_O = weightM.allocSpan(O->kn());
	// O->set_weight(W_O.data());
	// // const auto& W_UG = weightM.allocSpan(dual.Kn());
	// // UG->set_weight(W_UG.data());
	// auto W_U = weightM.allocSpan(dual.K*dual.N);
	// auto W_G = weightM.allocSpan(dual.K*dual.N);
	// auto vW_U = vortexWeight{(half*)W_U.data(), dual.N, dual.K};
	// auto vW_G = vortexWeight{(half*)W_G.data(), dual.N, dual.K};
	// dual.set_weight(vW_U, vW_G);
	// const auto& W_D = weightM.allocSpan(dual.K*dual.N);
	// D->set_weight(W_D.data());

	// const auto& W_LN_Attn = weightM.allocSpan(KQV->K);
	// layerNormAttention.setWeight(W_LN_Attn.data());
	// // const auto& W_LN_FFN = weightM.allocSpan(dual.K);
	// const auto& W_LN_FFN = weightM.allocSpan(dual.K);
	// layerNormFFN.setWeight(W_LN_FFN.data());
	// const auto& W_LN_Model = weightM.allocSpan(D->N);
	// layerNormModel.setWeight(W_LN_Model.data());
    // GEMV.setKVData(input_data->kv_data[layer]);
	// prefill.setKVData(input_data->kv_data[layer]);
	// roPEAppend.setKVData(input_data->kv_data[layer]);
	
	// const auto& W_LOGITS = weightM.allocSpan(LOGITS->kn());
	// LOGITS->set_weight(W_LOGITS.data());




	// spdlog::info("allocated {} halfs", weightM.getAllocation());

}

void NonOverlapLocalPipeline::ScheduleInit() {
	auto getMajor = [this](GEMM_NAME name, int x) {return getMajorType(config_data.gemm_op_tag[static_cast<int>(name)], x);};
		
	const int qdim = ModelConfig.model_head_dim *  ModelConfig.model_qo_heads_gpu;
	gemvOutput = tmpBufferM.allocTensor(config_data.global_batch_size, qdim, getMajor(GEMM_NAME::O, 2));

	const auto& O_output = tmpBufferM.allocTensor(O->M, O->N, PllmLayout::ROW_MAJOR);
	O->setA(gemvOutput);
	O->setD(O_output);
	
	const auto& LayerNormFFN_output = tmpBufferM.allocTensor(dual.M, dual.K, PllmLayout::ROW_MAJOR);
	layerNormFFN.setInput(O_output).setOutput(LayerNormFFN_output);
	// LN_FFN -> UG
	// UG->setA(layerNormFFN.getOutput()).setOutput(tmpBufferM.allocTensor(dual.M, dual.K, getMajor(GEMM_NAME::UG, 2)));
	
	const auto& Dual_output_0 = tmpBufferM.allocTensor(dual.M, dual.N, PllmLayout::ROW_MAJOR);
	const auto& Dual_output_1 = tmpBufferM.allocTensor(dual.M, dual.N, PllmLayout::ROW_MAJOR);
	const auto& activation_output = tmpBufferM.allocTensor(D->M, D->K, PllmLayout::ROW_MAJOR);
	dual.setA(layerNormFFN.getOutput());
	dual.setC(tmpBufferM.allocTensor(dual.M, dual.N, PllmLayout::ROW_MAJOR));
	dual.setD(Dual_output_0, Dual_output_1, activation_output);
	// UG -> activation
	// activation.setInput(UG->getD()).setOutput(tmpBufferM.allocTensor(D->M, D->K, getMajor(GEMM_NAME::D, 0)));
	// activation -> D
	const auto& D_output = tmpBufferM.allocTensor(D->M, D->N, PllmLayout::ROW_MAJOR);
	D->setA(activation_output);
	D->setC(O_output);
	D->setOutput(D_output);

	const auto& layerNormAttention_output = tmpBufferM.allocTensor(D->M, D->N, PllmLayout::ROW_MAJOR);
	layerNormAttention.setInput(D_output).setOutput(layerNormAttention_output);
	// LN_Attention -> O residual connection
	O->setC(D_output);
	KQV->setA(layerNormAttention.getOutput()).setOutput(tmpBufferM.allocTensor(KQV->M, KQV->N, getMajor(GEMM_NAME::KQV, 2)));
	

	gemvInput = tmpBufferM.allocTensor(config_data.global_batch_size, qdim, PllmLayout::ROW_MAJOR);

	

	const auto& keepTokenOutput = tmpBufferM.allocTensor(config_data.global_batch_size, D->N, PllmLayout::ROW_MAJOR);
	keepToken.setInput(tensor_cast<cutlass::half_t, half>(D_output)).setOutput(tensor_cast<cutlass::half_t, half>(keepTokenOutput));

	const auto& layerNormModel_output = tmpBufferM.allocTensor(config_data.global_batch_size, D->N, PllmLayout::ROW_MAJOR);
	layerNormModel.setInput(keepTokenOutput).setOutput(layerNormModel_output);

	const auto& LOGITS_output = tmpBufferM.allocTensor(config_data.global_batch_size, LOGITS->N, PllmLayout::ROW_MAJOR);
	LOGITS->setA(layerNormModel_output);
	LOGITS->setOutput(LOGITS_output);

	int* sample_output_alloc = (int*)tmpBufferM.alloc(config_data.global_batch_size * sizeof(int) / sizeof (half));
	pllmTensor<int> sample_output = {sample_output_alloc, config_data.global_batch_size, 1, PllmLayout::ROW_MAJOR};
	const auto& maxSampler_maxVals = tmpBufferM.allocTensor(config_data.global_batch_size, 1, PllmLayout::ROW_MAJOR);
	maxSampler.init(tensor_cast<cutlass::half_t, half>(LOGITS_output), 
					tensor_cast<cutlass::half_t, half>(maxSampler_maxVals), sample_output);

	genEmbedding.setOutput(tensor_cast<cutlass::half_t, half>(D->getD()));
	
	spdlog::info("allocated {} halfs", tmpBufferM.getAllocation());
}

void NonOverlapLocalPipeline::GEMMOpInit() {
	cutlass::half_t beta(1);
	setWeight(0);
	O->init(beta);
	// UG->init();
	dual.init();
	D->init(beta);
	KQV->init();
	for (int i = 0; i < gemmNum; ++i) {
		gemms[i]->setStream(stream_all);
	}

	LOGITS->set_weight(input_data->weight.lm_head); // important

	LOGITS->init();

	LOGITS->set_weight(input_data->weight.lm_head);

	dual.setStream(stream_all);
}

void NonOverlapLocalPipeline::GEMVOpUpdate() {
	spdlog::info("Update GEMV");

	uint32_t decode_batch_size = update_data.decodePrefillBorder;
	// const auto& [gemvDec_input, gemvPrefill_input] =
	// 	tensor_cast<cutlass::half_t, half>(gemvInput).splitTensor(getMajor(GEMM_NAME::O, 0),
	// 			  decode_batch_size,
	// 			  ModelConfig.max_batch_size - decode_batch_size);
	// const auto& [gemvDec_output, gemvPrefill_output] =
	// 	tensor_cast<cutlass::half_t, half>(gemvOutput).splitTensor(getMajor(GEMM_NAME::O, 0),
	// 			  decode_batch_size,
	// 			  ModelConfig.max_batch_size - decode_batch_size);



	uint32_t arr[] = {uint32_t(update_data.decodePrefillBorder), uint32_t(update_data.prefillNum)};
	std::span<uint32_t, 2> batch_sizes(arr, 2);
	auto total_batch_size = std::accumulate(batch_sizes.begin(), batch_sizes.end(), 0);
	assert (total_batch_size == update_data.decodePrefillBorder + update_data.prefillNum);
	const auto& kv_indptr_split = pllmTensor{update_data.kv_indptr, total_batch_size + 1}.splitTensor(
		PllmDimension::ROW, batch_sizes,/*overlap suffix*/ 1U);
	const auto& kv_last_page_len_split = pllmTensor{update_data.kv_last_page_len, total_batch_size}.
		splitTensor(PllmDimension::ROW, batch_sizes);
	const auto& input_ptr_split = pllmTensor{update_data.input_indptr, total_batch_size + 1}.splitTensor(
		PllmDimension::ROW, batch_sizes,/*overlap suffix*/ 1U);
	

	GEMV.init(decode_batch_size,
			  update_data.gemv_num_blocks[0],
			  input_ptr_split[0],
			  kv_indptr_split[0],
			  update_data.kv_indices,
			  kv_last_page_len_split[0],
			  tensor_cast<cutlass::half_t, half>(gemvInput),
			  tensor_cast<cutlass::half_t, half>(gemvOutput));
			//   gemvDec_input,
			//   gemvDec_output);
	log_tensor(spdlog::default_logger(), "GEMV kv_indptr_split", pllmTensor{update_data.kv_indptr, total_batch_size + 1}, 1, total_batch_size + 1);
	log_tensor(spdlog::default_logger(), "GEMV kv_last_page_len_split", pllmTensor{update_data.kv_last_page_len, total_batch_size}, 1, total_batch_size);
	log_tensor(spdlog::default_logger(), "GEMV input_ptr_split", pllmTensor{update_data.input_indptr, total_batch_size + 1}, 1, total_batch_size + 1);

	prefill.init(update_data.prefillNum,
				 108,
				 input_ptr_split[1],
				 kv_indptr_split[1],
				 update_data.kv_indices,
				 kv_last_page_len_split[1],
				 tensor_cast<cutlass::half_t, half>(gemvInput),
				 tensor_cast<cutlass::half_t, half>(gemvOutput));

	roPEAppend.update(update_data.decodePrefillBorder+update_data.prefillTokensNum, 
						tensor_cast<cutlass::half_t, half>(KQV->getD()), 
						tensor_cast<cutlass::half_t, half>(gemvInput), 
						pllmTensor<int>(update_data.rev_input_indptr, config_data.global_batch_size),
						pllmTensor<int>(update_data.per_token_offset, config_data.global_batch_size),
						pllmTensor<int>(update_data.kv_indices, ModelConfig.max_page_num),
						pllmTensor<int>(update_data.kv_indptr, config_data.global_batch_size + 1),
						pllmTensor<int>(update_data.kv_last_page_len, config_data.global_batch_size));
}

void NonOverlapLocalPipeline::OtherOpInit() {
	layerNormAttention.setStream(stream_all);
	layerNormFFN.setStream(stream_all);
	genEmbedding.setStream(stream_all);
	layerNormModel.setStream(stream_all);

	roPEAppend.setStream(stream_all);
	maxSampler.setStream(stream_all);
	keepToken.setStream(stream_all);

	// embedding one time set weight
	genEmbedding.setWeight(input_data->weight.embedding);
	layerNormModel.setWeight(input_data->weight.model_layernorm);

	// update event existance

}

void NonOverlapLocalPipeline::GEMVOpInit() {
	GEMV.setStream(stream_all);
	prefill.setStream(stream_all);
}

double NonOverlapLocalPipeline::totalCompute() {
	double total = 0;
	for(auto gemm : gemms)
		total += gemm->totalCompute();
	return total;
}