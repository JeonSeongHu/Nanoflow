#include "hip/hip_runtime.h"
#include "gemmFactory.cuh"
#include "pipeline.h"
#include "spdlog/spdlog.h"

NonOverlapNanoBatchPipeline::NonOverlapNanoBatchPipeline(vortexInitData* input_data,
									   int nrank,
									   int nranks,
									   int vnranks)
	: PipelineBase(input_data, nrank, nranks, vnranks) {
	// sampled tokens 
	hipHostMalloc(&outputTokens, 4096*sizeof(int));
	init();
}

void NonOverlapNanoBatchPipeline::init() {
	stream_all = hipStreamPerThread;
	spdlog::info("Init pipeline (non-overlap)");
#ifdef ENABLE_NETWORK
	spdlog::info("Init net");
	NetOpPrepare();
#endif
}

vortexOutputData NonOverlapNanoBatchPipeline::run() {
	constexpr bool enableGraph = false;
	spdlog::info("Start run");
	setWeight(0);

	if(!enableGraph)
		CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_START], stream_all));
	if(enableGraph) hipStreamBeginCapture(stream_all, hipStreamCaptureModeGlobal);

	constexpr int kNetNblocks = 64;

	// TODO: setup phase
	// layerNormPipeStart(stream_gemm);
	// KQV_START->run(stream_gemm);
	// GEMV_START(stream_gemm);


	for (int iter = 1; iter <= ModelConfig.run_layer*2; ++iter) {
		O->run();
		AG_O(stream_all, kNetNblocks, 1024, true);
		layerNormFFN.run();
		UG->run();
		activation.run();
		D->run();
		AR_D(stream_all, kNetNblocks, 1024, true);
		layerNormAttention.run();

		if (iter == ModelConfig.run_layer*2) break;
		setWeight(iter%5);
		KQV->run();
		roPEAppend.run();
		GEMV.run();
		O_TR.skip();
		AG_GEMV(stream_all, kNetNblocks, 1024, true);

		if (update_data.prefillNum /2 > 0)
			prefill.run();
	}
	hipMemcpyAsync(
		outputTokens, input_data->tmp_buffer, 2048 * sizeof(int), hipMemcpyDeviceToHost, stream_all);
	// // End capture
	hipGraph_t graph;
	if (enableGraph) {
		hipStreamEndCapture(stream_all, &graph);
		if (graph == NULL) {
			spdlog::error("Failed to create graph");
			exit(1);
		}
		spdlog::info("Graph created");
		hipGraphExec_t instance;
		hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
		spdlog::info("Graph instantiated");
		CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_START], stream_all));
		for(int i = 0; i < 10; i ++)
			hipGraphLaunch(instance, stream_all);
		spdlog::info("Graph launched");
	}

	// Record an event when the GEMMs are complete
	CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_END], stream_all));

	// Wait for work on the device to complete.
	CUDA_CHECK(hipEventSynchronize(events[EventManager::GEMM_TIMING_END]));

	// Measure elapsed runtime
	float runtime_ms = 0;
	CUDA_CHECK(hipEventElapsedTime(&runtime_ms,
									events[EventManager::GEMM_TIMING_START],
									events[EventManager::GEMM_TIMING_END]));
	// Compute average runtime and GFLOPs.
	runtime_ms = double(runtime_ms);
	double gflops = totalCompute() / runtime_ms / 1e6;
	double bandwidth = sizeof(__half) * (96731136 + 1.25 / 80 * 160 * 1024 * 1024 * 1024 / 2) /
					   (runtime_ms / 1000) / (1 << 30);
	spdlog::info("Total running cost (ms) of one microbatch is {}", runtime_ms);

	vortexOutputData d;
	d.global_batch_size = 0;
	d.sampled_tokens = 0;

	return d;
}

void NonOverlapNanoBatchPipeline::update(vortexUpdateData* update_data) {
	this->update_data = *update_data;
	GEMVOpUpdate();
	// log first batch size
	spdlog::info(
		"prefill: {}, decode: {}", update_data->prefillNum, update_data->decodePrefillBorder);
	spdlog::info("Batch size: {}, {}, {}, {}",
				 update_data->gemv_batch_size[0],
				 update_data->gemv_batch_size[1],
				 update_data->gemv_batch_size[2],
				 update_data->gemv_batch_size[3]);
}

void NonOverlapNanoBatchPipeline::config(vortexConfigData* config_data) {
	spdlog::info("Config non-overlap pipeline");
    for (int i = 0; i < gemmNum; ++i) {
        gemms[i] = generateGEMM(gemmConfig[i]);
    }
	this->config_data = *config_data;
    int globalbatch = config_data->global_batch_size / 2;
    O ->set_shape(globalbatch, ModelConfig.model_hidden_dim_pergpu, ModelConfig.model_hidden_dim);
    UG ->set_shape(globalbatch, ModelConfig.model_ff_dim_gpu + ModelConfig.model_ff_dim_gpu, ModelConfig.model_hidden_dim);
    D ->set_shape(globalbatch, ModelConfig.model_hidden_dim, ModelConfig.model_ff_dim_gpu);
	KQV->set_shape(globalbatch,
				   (ModelConfig.model_kv_heads_gpu + ModelConfig.model_kv_heads_gpu + ModelConfig.model_qo_heads_gpu) * ModelConfig.model_head_dim,
				   ModelConfig.model_hidden_dim);
	KQV_START->set_shape(globalbatch,
						 (ModelConfig.model_kv_heads_gpu + ModelConfig.model_kv_heads_gpu + ModelConfig.model_qo_heads_gpu) * ModelConfig.model_head_dim,
						 ModelConfig.model_hidden_dim);
	spdlog::info("Init schedule");
	ScheduleInit();
	spdlog::info("Init gemm");
	GEMMOpInit();
	spdlog::info("Init other");
	OtherOpInit();
}

void NonOverlapNanoBatchPipeline::setWeight(int layer) {
    AllocationManager<cutlass::half_t> weightM(ptr_cast<cutlass::half_t>(input_data->tmp_buffer), input_data->weight_size);
	const auto& W_O = weightM.allocSpan(O->kn());
	O->set_weight(W_O.data());
	const auto& W_UG = weightM.allocSpan(UG->kn());
	UG->set_weight(W_UG.data());
	const auto& W_D = weightM.allocSpan(UG->kn());
	D->set_weight(W_D.data());
	const auto& W_KQV = weightM.allocSpan(KQV->kn());
	KQV->set_weight(W_KQV.data());
	const auto& W_LN_Attn = weightM.allocSpan(KQV->K);
	layerNormAttention.setWeight(W_LN_Attn.data());
	const auto& W_LN_FFN = weightM.allocSpan(UG->K);
	layerNormFFN.setWeight(W_LN_FFN.data());
    GEMV.setKVData(input_data->kv_data[layer]);
	prefill.setKVData(input_data->kv_data[layer]);
}

void NonOverlapNanoBatchPipeline::ScheduleInit() {
	auto getMajor = [this](GEMM_NAME name, int x) {return getMajorType(config_data.gemm_op_tag[static_cast<int>(name)], x);};
	auto getDim = [this, getMajor](GEMM_NAME name, int x) {return static_cast<PllmDimension>(getMajor(name, x));};
	// (prev)AG_GEMV -> O (col major)
	AG_GEMV.init(comm, connections, rank, nranks, tmpBufferM.allocTensor(O->M, O->K, getMajor(GEMM_NAME::O, 0)));
	O->setA(AG_GEMV.getOutput());
	// O (col major) -> AG_O
	AG_O.init(comm, connections, rank, nranks, tmpBufferM.allocTensor(UG->M, UG->K, getMajor(GEMM_NAME::O, 2)));
	O->setD(AG_O.getInput().getSubTensor(rank, vnranks, getDim(GEMM_NAME::O, 2)));
	// AG_O -> LN_FFN
	layerNormFFN.setInput(AG_O.getOutput()).setOutput(tmpBufferM.allocTensor(UG->M, UG->K, getMajor(GEMM_NAME::UG, 0)));
	// LN_FFN -> UG
	UG->setA(layerNormFFN.getOutput()).setOutput(tmpBufferM.allocTensor(UG->M, UG->K, getMajor(GEMM_NAME::UG, 2)));
	// UG -> activation
	activation.setInput(UG->getD()).setOutput(tmpBufferM.allocTensor(D->M, D->K, getMajor(GEMM_NAME::D, 0)));
	// activation -> D
	D->setA(activation.getOutput());
	// D -> AR_D
	AR_D.init(comm, connections, rank, nranks, tmpBufferM.allocTensor(D->M, D->N, getMajor(GEMM_NAME::D, 2)), tmpBufferM.allocTensor(D->M, D->N, getMajor(GEMM_NAME::D, 2)));
	D->setOutput(AR_D.getInput());
	// AR_D -> LN_Attention
	layerNormAttention.setInput(AR_D.getOutput()).setOutput(tmpBufferM.allocTensor(D->M, D->N, getMajor(GEMM_NAME::O, 2)));
	// LN_Attention -> O residual connection
	O->setC(layerNormAttention.getOutput().getSubTensor(rank, vnranks, getDim(GEMM_NAME::D, 2)));
	KQV->setA(layerNormAttention.getOutput()).setOutput(tmpBufferM.allocTensor(KQV->M, KQV->N, getMajor(GEMM_NAME::KQV, 2)));
	gemvInput = tmpBufferM.allocTensor(ModelConfig.max_batch_size, KQV->N, PllmLayout::ROW_MAJOR);
	gemvOutput = tmpBufferM.allocTensor(ModelConfig.max_batch_size,KQV->N, PllmLayout::ROW_MAJOR);
	// TODO: ropeAppend is not implemented yet
	O_TR.setInput(gemvOutput).setOutput(AG_GEMV.getInput().getSubTensor(rank, vnranks, static_cast<PllmDimension>(!((int)getDim(GEMM_NAME::O, 2)))));

}

void NonOverlapNanoBatchPipeline::GEMMOpInit() {
	cutlass::half_t beta(1);
	setWeight(0);
	O->init(beta);
	UG->init();
	D->init();
	KQV->init();
	for (int i = 0; i < gemmNum; ++i) {
		gemms[i]->setStream(stream_all);
	}

	layerNormAttention.setStream(stream_all);
	layerNormFFN.setStream(stream_all);
	GEMV.setStream(stream_all);
	prefill.setStream(stream_all);
	O_TR.setStream(stream_all);
	activation.setStream(stream_all);
	roPEAppend.setStream(stream_all);
}

void NonOverlapNanoBatchPipeline::GEMVOpUpdate() {
	auto getMajor = [this](GEMM_NAME name, int x) {return static_cast<PllmDimension>((config_data.gemm_op_tag[static_cast<int>(name)], x));};
	spdlog::info("Update GEMV");

	uint32_t decode_batch_size = update_data.decodePrefillBorder;
	const auto& [gemvDec_input, gemvPrefill_input] =
		tensor_cast<cutlass::half_t, half>(gemvInput).splitTensor(getMajor(GEMM_NAME::O, 0),
				  decode_batch_size,
				  ModelConfig.max_batch_size - decode_batch_size);
	const auto& [gemvDec_output, gemvPrefill_output] =
		tensor_cast<cutlass::half_t, half>(gemvOutput).splitTensor(getMajor(GEMM_NAME::O, 0),
				  decode_batch_size,
				  ModelConfig.max_batch_size - decode_batch_size);
	spdlog::info("Decode batch size: {}", decode_batch_size);

	
	uint32_t arr[] = {uint32_t(decode_batch_size), uint32_t(update_data.prefillNum)};
	std::span<uint32_t, 2> batch_sizes(arr, 2);
	spdlog::info("Batch sizes: {}, {}", batch_sizes[0], batch_sizes[1]);
	auto total_batch_size = std::accumulate(batch_sizes.begin(), batch_sizes.end(), 0);
	assert (total_batch_size == update_data.decodePrefillBorder + update_data.prefillNum);
	const auto& kv_indptr_split = pllmTensor{update_data.kv_indptr, total_batch_size + 1}.splitTensor(
		PllmDimension::ROW, batch_sizes,/*overlap suffix*/ 1U);
	const auto& kv_last_page_len_split = pllmTensor{update_data.kv_last_page_len, total_batch_size}.
		splitTensor(PllmDimension::ROW, batch_sizes);
	const auto& input_ptr_split = pllmTensor{update_data.input_indptr, total_batch_size + 1}.splitTensor(
		PllmDimension::ROW, batch_sizes,/*overlap suffix*/ 1U);
	spdlog::info("KV indptr split: {}, {}", kv_indptr_split[0].size(), kv_indptr_split[1].size());
	decode_batch_size = update_data.decodePrefillBorder/2;
	GEMV.init(decode_batch_size,
			  update_data.gemv_num_blocks[0],
			  input_ptr_split[0],
			  kv_indptr_split[0],
			  update_data.kv_indices,
			  kv_last_page_len_split[0],
			  gemvDec_input,
			  gemvDec_output);
	spdlog::info("Decode {}", decode_batch_size);
	prefill.init(update_data.prefillNum/2,
				 40,
				 input_ptr_split[1],
				 kv_indptr_split[1],
				 update_data.kv_indices,
				 kv_last_page_len_split[1],
				 tensor_cast<cutlass::half_t, half>(gemvInput),
				 tensor_cast<cutlass::half_t, half>(gemvOutput));
	spdlog::info("Prefill {}", update_data.prefillNum);
}

double NonOverlapNanoBatchPipeline::totalCompute() {
	double total = 0;
	for(auto gemm : gemms)
		total += gemm->totalCompute();
	return total;
}